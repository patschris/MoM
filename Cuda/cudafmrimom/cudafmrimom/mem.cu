#include "hip/hip_runtime.h"
///////////////////////////////////////////////
//              File: mem.cu                 //
///////////////////////////////////////////////
#include <iostream>
#include <ctime>
#include <hiprand.h>
#include <hipsolver.h>
#include "hipblas.h"
#include "declarations.h"
#include "errorTypes.h"
#include "Globals.h"

///////////////////////////////////////////////////////////////////////////////
/* Global variables */
extern int *devInfo;
extern hipblasHandle_t handle;
extern hiprandGenerator_t generator;
extern hipsolverHandle_t cusolverH;
extern datatype *I, *KK, *KK2, *KVOXELS, *KVOXELS2, *TIMECOMPK, *TIMECOMPK2, *Kv, *TIMECOMPVOXELS, *X, *D, *S, *W, *d_W;
///////////////////////////////////////////////////////////////////////////////


/* Free all the previously allocated memory */
void cleanup() {
	hipError_t err;
	hiprandStatus_t curstat;
	hipblasStatus_t cubstat;
	hipsolverStatus_t cusstat;
	if (W) free(W);
	if (generator && (curstat = hiprandDestroyGenerator(generator)) != HIPRAND_STATUS_SUCCESS)
		printf("hiprandDestroyGenerator failed: %s\n", curandGetErrorString(curstat));
	if (handle && (cubstat = hipblasDestroy(handle)) != HIPBLAS_STATUS_SUCCESS)
		printf("hipblasDestroy failed: %s\n", cublasGetErrorString(cubstat));
	if (X && (err = hipFree(X)) != hipSuccess) printf("hipFree X failed: %s\n", hipGetErrorString(err));
	if (D && (err = hipFree(D)) != hipSuccess) printf("hipFree D failed: %s\n", hipGetErrorString(err));
	if (S && (err = hipFree(S)) != hipSuccess) printf("hipFree S failed: %s\n", hipGetErrorString(err));
	if (I && (err = hipFree(I)) != hipSuccess) printf("hipFree cuI failed: %s\n", hipGetErrorString(err));
	if (KK && (err = hipFree(KK)) != hipSuccess) printf("hipFree KK failed: %s\n", hipGetErrorString(err));
	if (KK2 && (err = hipFree(KK2)) != hipSuccess) printf("hipFree KK2 failed: %s\n", hipGetErrorString(err));
	if (KVOXELS && (err = hipFree(KVOXELS)) != hipSuccess) printf("hipFree KVOXELS failed: %s\n", hipGetErrorString(err));
	if (KVOXELS2 && (err = hipFree(KVOXELS2)) != hipSuccess) printf("hipFree KVOXELS2 failed: %s\n", hipGetErrorString(err));
	if (TIMECOMPK && (err = hipFree(TIMECOMPK)) != hipSuccess) printf("hipFree TIMECOMPK failed: %s\n", hipGetErrorString(err));
	if (TIMECOMPK2 && (err = hipFree(TIMECOMPK2)) != hipSuccess) printf("hipFree TIMECOMPK2 failed: %s\n", hipGetErrorString(err));
	if (Kv && (err = hipFree(Kv)) != hipSuccess) printf("hipFree Kv failed: %s\n", hipGetErrorString(err));
	if (TIMECOMPVOXELS && (err = hipFree(TIMECOMPVOXELS)) != hipSuccess) printf("hipFree TIMECOMPVOXELS failed: %s\n", hipGetErrorString(err));
	if (devInfo && (err = hipFree(devInfo)) != hipSuccess) printf("hipFree devInfo failed: %s\n", hipGetErrorString(err));
	if (cusolverH && (cusstat = hipsolverDnDestroy(cusolverH)) != HIPSOLVER_STATUS_SUCCESS)
		printf("hipsolverDnDestroy failed: %s\n", cusolverGetErrorString(cusstat));
	if (d_W && (err = hipFree(d_W)) != hipSuccess) printf("hipFree d_W failed: %s\n", hipGetErrorString(err));
	if (hipDeviceReset() != hipSuccess) printf("hipDeviceReset failed: %s\n", hipGetErrorString(err));
}


/* Allocate all necessary memory for global variables */
void allocate() {
	hipError_t err;
	hipblasStatus_t cubStat;
	hiprandStatus_t curStat;
	hipsolverStatus_t cusStat;
	if ((W = (datatype *)malloc(K * sizeof(datatype))) == NULL) {
		perror("malloc failed W");
		cleanup();
		exit(-1);
	}
	if ((cubStat = hipblasCreate(&handle)) != HIPBLAS_STATUS_SUCCESS) {
		printf("hipblasCreate failed: %s\n", cublasGetErrorString(cubStat));
		cleanup();
		exit(-1);
	}
	if ((cusStat = hipsolverDnCreate(&cusolverH)) != HIPSOLVER_STATUS_SUCCESS) {
		printf("hipsolverDnCreate failed: %s\n", cusolverGetErrorString(cusStat));
		cleanup();
		exit(-1);
	}
	if ((curStat = hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT)) != HIPRAND_STATUS_SUCCESS) {
		printf("hiprandCreateGenerator failed: %s\n", curandGetErrorString(curStat));
		cleanup();
		exit(-1);
	}
	if ((curStat = hiprandSetPseudoRandomGeneratorSeed(generator, (unsigned long long) clock())) != HIPRAND_STATUS_SUCCESS) {
		printf("hiprandSetPseudoRandomGeneratorSeed failed: %s\n", curandGetErrorString(curStat));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&X, Globals::rowsX * Globals::colsX * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed cuX: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&D, Globals::rowsX * K * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed D: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&S, K * Globals::colsX * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed S: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&I, K * K * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed cuI: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&KK, K * K * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed KK: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&KK2, K * K * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed KK2: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&KVOXELS, K * Globals::colsX * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed KVOXELS: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&KVOXELS2, K * Globals::colsX * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed KVOXELS: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&TIMECOMPK, Globals::rowsX * K * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed TIMECOMPK: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&TIMECOMPK2, Globals::rowsX * K * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed TIMECOMPK2: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&Kv, K * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed Kv: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void **)&TIMECOMPVOXELS, Globals::rowsX * Globals::colsX * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed TIMECOMPVOXELS: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void**)&d_W, K * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc failed d_W: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if ((err = hipMalloc((void**)&devInfo, sizeof(int))) != hipSuccess) {
		printf("hipMalloc failed devInfo: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
}