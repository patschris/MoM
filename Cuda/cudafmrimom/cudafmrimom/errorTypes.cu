///////////////////////////////////////////////
//          File: errorTypes.cu              //
///////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hipsolver.h>
#include "hipblas.h"


/* Finds the proper error string based on the error code of a library's curand function */
const char *curandGetErrorString(hiprandStatus_t error) {
	switch (error) {
		case HIPRAND_STATUS_SUCCESS: return "CURAND_STATUS_SUCCESS";
		case HIPRAND_STATUS_VERSION_MISMATCH: return "CURAND_STATUS_VERSION_MISMATCH";
		case HIPRAND_STATUS_NOT_INITIALIZED: return "CURAND_STATUS_NOT_INITIALIZED";
		case HIPRAND_STATUS_ALLOCATION_FAILED: return "CURAND_STATUS_ALLOCATION_FAILED";
		case HIPRAND_STATUS_TYPE_ERROR: return "CURAND_STATUS_TYPE_ERROR";
		case HIPRAND_STATUS_OUT_OF_RANGE: return "CURAND_STATUS_OUT_OF_RANGE";
		case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE: return "CURAND_STATUS_LENGTH_NOT_MULTIPLE";
		case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED: return "CURAND_STATUS_DOUBLE_PRECISION_REQUIRED";
		case HIPRAND_STATUS_LAUNCH_FAILURE: return "CURAND_STATUS_LAUNCH_FAILURE";
		case HIPRAND_STATUS_PREEXISTING_FAILURE: return "CURAND_STATUS_PREEXISTING_FAILURE";
		case HIPRAND_STATUS_INITIALIZATION_FAILED: return "CURAND_STATUS_INITIALIZATION_FAILED";
		case HIPRAND_STATUS_ARCH_MISMATCH: return "CURAND_STATUS_ARCH_MISMATCH";
		case HIPRAND_STATUS_INTERNAL_ERROR: return "CURAND_STATUS_INTERNAL_ERROR";
	}
	return "UNKNOWN ERROR";
}


/* Finds the proper error string based on the error code of a library's cublas function */
const char *cublasGetErrorString(hipblasStatus_t status) {
	switch (status) {
		case HIPBLAS_STATUS_SUCCESS: return "CUBLAS_STATUS_SUCCESS";
		case HIPBLAS_STATUS_NOT_INITIALIZED: return "CUBLAS_STATUS_NOT_INITIALIZED";
		case HIPBLAS_STATUS_ALLOC_FAILED: return "CUBLAS_STATUS_ALLOC_FAILED";
		case HIPBLAS_STATUS_INVALID_VALUE: return "CUBLAS_STATUS_INVALID_VALUE";
		case HIPBLAS_STATUS_ARCH_MISMATCH: return "CUBLAS_STATUS_ARCH_MISMATCH";
		case HIPBLAS_STATUS_MAPPING_ERROR: return "CUBLAS_STATUS_MAPPING_ERROR";
		case HIPBLAS_STATUS_EXECUTION_FAILED: return "CUBLAS_STATUS_EXECUTION_FAILED";
		case HIPBLAS_STATUS_INTERNAL_ERROR: return "CUBLAS_STATUS_INTERNAL_ERROR";
	}
	return "UNKNOWN ERROR";
}


/* Finds the proper error string based on the error code of a library's cusolver function */
const char *cusolverGetErrorString(hipsolverStatus_t status) {
	switch (status) {
		case HIPSOLVER_STATUS_SUCCESS: return "CUSOLVER_STATUS_SUCCESS";
		case HIPSOLVER_STATUS_NOT_INITIALIZED: return "CUSOLVER_STATUS_NOT_INITIALIZED";
		case HIPSOLVER_STATUS_ALLOC_FAILED: return "CUSOLVER_STATUS_ALLOC_FAILED";
		case HIPSOLVER_STATUS_INVALID_VALUE: return "CUSOLVER_STATUS_INVALID_VALUE";
		case HIPSOLVER_STATUS_ARCH_MISMATCH: return "CUSOLVER_STATUS_ARCH_MISMATCH";
		case HIPSOLVER_STATUS_EXECUTION_FAILED: return "CUSOLVER_STATUS_execution failed";
		case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "CUSOLVER_STATUS_Matrix not supported";
		case HIPSOLVER_STATUS_INTERNAL_ERROR: return "CUSOLVER_STATUS_INTERNAL_ERROR";
	}
	return "UNKNOWN ERROR";
}