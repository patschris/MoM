#include "hip/hip_runtime.h"
///////////////////////////////////////////////
//            File: cuLibs.cu                //
///////////////////////////////////////////////
#include <iostream>
#include <hipsolver.h>
#include <hiprand.h>
#include "hipblas.h"
#include "declarations.h"
#include "errorTypes.h"
#include "mem.h"
#include INCLUDE_FILE(MATLAB_include,mat.h)
#include INCLUDE_FILE(MATLAB_include,matrix.h)
#include "primaryFunctions.h"


/* Calculates the frobenius norm of the given array */
datatype frobeniusNorm(hipblasHandle_t handle, datatype *Array, int size) {
	datatype retval;
	hipblasStatus_t stat;
	#if (MODE)
		if ((stat = hipblasDnrm2(handle, size, Array, 1, &retval)) != HIPBLAS_STATUS_SUCCESS) {
			printf("cublasDnmr2 failed: %s\n", cublasGetErrorString(stat));
			cleanup();
			exit(-1);
		}
	#else
		if ((stat = hipblasSnrm2(handle, size, Array, 1, &retval)) != HIPBLAS_STATUS_SUCCESS) {
			printf("cublasSnmr2 failed: %s\n", cublasGetErrorString(stat));
			cleanup();
			exit(-1);
		}
	#endif
	return retval;
}


/* Generate numbers using normal distribution
   http://docs.nvidia.com/cuda/hiprand/host-api-overview.html#axzz4SrFExYZh */
void randn(hiprandGenerator_t generator, datatype *Arr, int size) {
	hiprandStatus_t stat;
	#if (MODE)
		if ((stat = hiprandGenerateNormalDouble(generator, Arr, size, (datatype)0, (datatype)1)) != HIPRAND_STATUS_SUCCESS) {
			printf("hiprandGenerateNormalDouble failed: %s\n", curandGetErrorString(stat));
			cleanup();
			exit(-1);
		}
	#else
		if ((stat = hiprandGenerateNormal(generator, Arr, size, (datatype)0, (datatype)1)) != HIPRAND_STATUS_SUCCESS) {
			printf("hiprandGenerateNormal failed: %s\n", curandGetErrorString(stat));
			cleanup();
			exit(-1);
		}
	#endif
}


/* Returns the square root of the maximum eigenvalue of the given array using power method */
datatype findMaxSqrtEigenvalue(hipsolverHandle_t cusolverH, datatype *dA, datatype *W, datatype *d_W, int *devInfo, const int m) {
	int lwork = 0;
	hipError_t err;
	const int lda = m;
	datatype *d_work = 0;
	hipsolverStatus_t stat;
	#if (MODE)
		if ((stat = hipsolverDnDsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER, m, dA, lda, d_W, &lwork)) != HIPSOLVER_STATUS_SUCCESS) {
			printf("hipsolverDnDsyevd_bufferSize failed: %s\n", cusolverGetErrorString(stat));
			cleanup();
			exit(-1);
		}
	#else
		if ((stat = hipsolverDnSsyevd_bufferSize(cusolverH, HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_LOWER, m, dA, lda, d_W, &lwork)) != HIPSOLVER_STATUS_SUCCESS) {
			printf("hipsolverDnSsyevd_bufferSize failed: %s\n", cusolverGetErrorString(stat));
			cleanup();
			exit(-1);
		}
	#endif
	if ((err = hipMalloc((void**)&d_work, lwork * sizeof(datatype))) != hipSuccess) {
		printf("hipMalloc d_work failed: %s\n", hipGetErrorString(err));
		if (d_work && (err = hipFree(d_work)) != hipSuccess) printf("hipFree d_work: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	#if (MODE)
		if ((stat = hipsolverDnDsyevd(cusolverH, HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_UPPER, m, dA, lda, d_W, d_work, lwork, devInfo)) != HIPSOLVER_STATUS_SUCCESS) {
			printf("hipsolverDnDsyevd failed: %s\n", cusolverGetErrorString(stat));
			if (d_work && (err = hipFree(d_work)) != hipSuccess) printf("hipFree d_work: %s\n", hipGetErrorString(err));
			cleanup();
			exit(-1);
		}
	#else
		if ((stat = hipsolverDnSsyevd(cusolverH, HIPSOLVER_EIG_MODE_NOVECTOR, HIPBLAS_FILL_MODE_UPPER, m, dA, lda, d_W, d_work, lwork, devInfo)) != HIPSOLVER_STATUS_SUCCESS) {
			printf("hipsolverDnSsyevd failed: %s\n", cusolverGetErrorString(stat));
			if (d_work && (err = hipFree(d_work)) != hipSuccess) printf("hipFree d_work: %s\n", hipGetErrorString(err));
			cleanup();
			exit(-1);
		}
	#endif	
	if ((err = hipMemcpy(W, d_W, m * sizeof(datatype), hipMemcpyDeviceToHost)) != hipSuccess) {
		printf("hipMemcpy W failed: %s\n", hipGetErrorString(err));
		if (d_work && (err = hipFree(d_work)) != hipSuccess) printf("hipFree d_work: %s\n", hipGetErrorString(err));
		cleanup();
		exit(-1);
	}
	if (d_work && (err = hipFree(d_work)) != hipSuccess) printf("hipFree d_work: %s\n", hipGetErrorString(err));
	return sqrt(W[m - 1]);
}


/* Multiplies two matrices (category=0:C=A*B, category=1:C=A'*B, category=2:C=A*B') */
void gpu_blas_mmul(hipblasHandle_t handle, const datatype *A, const datatype *B, datatype *C,
	int category, const int rowA, const int colA, const int rowB, const int colB) {

	int lda, ldb, ldc;
	const datatype alf = 1;
	const datatype bet = 0;
	const datatype *alpha = &alf;
	const datatype *beta = &bet;
	hipblasStatus_t status;
	if (category == 0) { // C=A*B ( http://peterwittek.com/cublas-matrix-c-style.html )
		lda = colB; ldb = colA; ldc = colB;
		if (colA != rowB) {
			printf("colA != rowB. Cannot multiply the matrices\n");
			cleanup();
			exit(-1);
		}
		#if (MODE)
			status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, colB, rowA, colA, alpha, B, lda, A, ldb, beta, C, ldc);
		#else 
			status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, colB, rowA, colA, alpha, B, lda, A, ldb, beta, C, ldc);
		#endif
	}
	else if (category == 1) { // C = A'*B ( http://stackoverflow.com/questions/14595750/transpose-matrix-multiplication-in-cublas-howto )
		lda = colA; ldb = colB; ldc = colB;
		if (rowA != rowB) {
			printf("rowA != rowB. Cannot multiply the matrices\n");
			cleanup();
			exit(-1);
		}
		#if (MODE)
			status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, colB, colA, rowB, alpha, B, ldb, A, lda, beta, C, ldc);
		#else
			status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, colB, colA, rowB, alpha, B, ldb, A, lda, beta, C, ldc);
		#endif
	}
	else if (category == 2) { // C=A*B' 
		lda = colA; ldb = colB; ldc = rowB;
		if (colA != colB) {
			printf("colA != colB. Cannot multiply the matrices\n");
			cleanup();
			exit(-1);
		}
		#if (MODE)
			status = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rowB, rowA, colB, alpha, B, ldb, A, lda, beta, C, ldc);
		#else
			status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, rowB, rowA, colB, alpha, B, ldb, A, lda, beta, C, ldc);
		#endif
	}
	else {
		printf("gpu_blas_mmul : not valid status %d \n", category);
		cleanup();
		exit(-1);
	}
	if (status != hipSuccess) {
		printf("Cublas multiplication failed: %s\n", cublasGetErrorString(status));
		cleanup();
		exit(-1);
	}
}